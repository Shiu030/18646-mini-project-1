#include "hip/hip_runtime.h"
/*
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
#define BLOCK_WIDTH 2

namespace cuda
{
  __global__ void matrix_mul_kernel(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, int sq_dimension)
  {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float temp_result = 0;
    
    for(int k = 0; k < sq_dimension; k++)
    {	
	  temp_result += sq_matrix_1[row * sq_dimension + k] * sq_matrix_2[k * sq_dimension + col];
    }

    sq_matrix_result[row * sq_dimension + col] += temp_result;
  }
  
  void matrix_multiplication(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, unsigned int sq_dimension)
  {
    int size = sq_dimension * sq_dimension * sizeof(float);
    float *sq_matrix_1_d, *sq_matrix_2_d, *sq_matrix_result_d;
    
    /***************************************************
    Step 1: Allocation of memory on device memory  
    ****************************************************/
    
    /* copy sq_matrix_1 and sq_matrix_2 to device memory */
    hipMalloc((void**) &sq_matrix_1_d, size);
    hipMemcpy(sq_matrix_1_d, sq_matrix_1, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &sq_matrix_2_d, size);
    hipMemcpy(sq_matrix_2_d, sq_matrix_2, size, hipMemcpyHostToDevice);
    
    /*allocate sq_matrix_result on host */
    hipMalloc((void**) &sq_matrix_result_d, size);
    
    /***************************************************
    Step 2: Invoke kernel 
    ****************************************************/
    short adjusted_block_width = 16;
    if (sq_dimension % 15 == 0) adjusted_block_width = 15;
    else if (sq_dimension % 14 == 0) adjusted_block_width = 14;
    else if (sq_dimension % 13 == 0) adjusted_block_width = 13;
    else if (sq_dimension % 12 == 0) adjusted_block_width = 12;
    else if (sq_dimension % 11 == 0) adjusted_block_width = 11;
    else if (sq_dimension % 10 == 0) adjusted_block_width = 10;
    else if (sq_dimension % 9 == 0) adjusted_block_width = 9;
    else if (sq_dimension % 8 == 0) adjusted_block_width = 8;
    else if (sq_dimension % 7 == 0) adjusted_block_width = 7;
    else if (sq_dimension % 6 == 0) adjusted_block_width = 6;
    else if (sq_dimension % 5 == 0) adjusted_block_width = 5;
    else if (sq_dimension % 4 == 0) adjusted_block_width = 4;
    else if (sq_dimension % 3 == 0) adjusted_block_width = 3;
    else if (sq_dimension % 2 == 0) adjusted_block_width = 2;
    else adjusted_block_width = 1;

    int blockNum = ceil(sq_dimension * 1.0 / adjusted_block_width);
    dim3 dimBlock(adjusted_block_width, adjusted_block_width);
    dim3 dimGrid(blockNum, blockNum);

    matrix_mul_kernel<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
    
    /***************************************************
    Step 3: Transfer result from device to host 
    ****************************************************/
    hipMemcpy(sq_matrix_result, sq_matrix_result_d, size, hipMemcpyDeviceToHost);
    hipFree(sq_matrix_1_d);
    hipFree(sq_matrix_2_d);
    hipFree(sq_matrix_result_d);
  }  
} // namespace cuda
